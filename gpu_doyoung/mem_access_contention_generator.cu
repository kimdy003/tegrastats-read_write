#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/syscall.h>
#include <unistd.h>
#include <sys/types.h>
#include <nvToolsExt.h>
#include <string.h>
#include <assert.h>
#include <sys/time.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
//#include <helper_functions.h>
//#include <hip/hip_runtime_api.h>

	__global__
void vertorADDGPU(unsigned int n, float *x, float *y, float *z) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	// Memory reads of 8MB and memory writes of 4MB in turn
	// because n = 0x1 << 20
//	if(idx < n) {
		z[idx] = x[idx] + y[idx];
//	}
}

double time_is_it_now(){
	struct timeval time;
	if(gettimeofday(&time,NULL)){
			return 0;
	}
	return (double)time.tv_sec + (double)time.tv_usec*.000001;
}

int main(int argc, char *argv[]) {
	unsigned ;int n = 0x1 << 20;
	int thr_num;
	float *x, *y, *z, *d_x, *d_y, *d_z;
	unsigned int flags = hipHostMallocMapped;

	// Check the number of arguments
	if (argc != 2) {
		printf("please use with one argument\n");
		printf("The argument is thread num\n");
		return 1;
	}
	thr_num = atoi(argv[1]);

	// Dynamic memory allocation by using zero-copy memory mechanism
#if 0
	x = (float*)malloc(n*sizeof(float));
	y = (float*)malloc(n*sizeof(float));
	z = (float*)malloc(n*sizeof(float));
#else
	hipHostAlloc((void **)&x, n * sizeof(float), flags);
	hipHostAlloc((void **)&y, n * sizeof(float), flags);
	hipHostAlloc((void **)&z, n * sizeof(float), flags);

#endif
	// Initialize the two input arrays
	for (unsigned int i = 0; i < n; i++) {
        x[i] = rand() / (float) RAND_MAX;
        y[i] = rand() / (float) RAND_MAX;
	}
#if 0
	hipMalloc((void**)&d_x, n*sizeof(float));
	hipMalloc((void**)&d_y, n*sizeof(float));
	hipMalloc((void**)&d_z, n*sizeof(float));
	hipMemcpy(d_x, x, n*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_y, y, n*sizeof(float), hipMemcpyHostToDevice);
#else
	hipHostGetDevicePointer((void **)&d_x, (void *)x, 0);
	hipHostGetDevicePointer((void **)&d_y, (void *)y, 0);
	hipHostGetDevicePointer((void **)&d_z, (void *)z, 0);
#endif
	// Set the number of threads
	dim3 block(thr_num);
	dim3 grid((unsigned int)ceil(n / (float)block.x));
	
	//double start;
	//int cnt =0;
	//start = time_is_it_now();
	while(1) {
		//printf("start\n");
		vertorADDGPU<<<thr_num, 1024>>>(n, d_x, d_y, d_z);
		//hipDeviceSynchronize();
		//usleep(500);
#if 0
		if(cnt%10000 == 0){
			printf(" execution time : %lf \n", time_is_it_now() - start);
			start = time_is_it_now();
		}
#endif		
	//	cnt++;
	}
#if 0
	checkCudaErrors(hipDeviceSynchronize());
#else
	hipDeviceSynchronize();
#endif
	hipFree(d_x);
	hipFree(d_y);
	return 1;
}
